/***************************************************************************
 *
 * Paralell version of Gauss-Jordan row reduction
 *
 ***************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_SIZE 4096
#define THREADS_PER_BLOCK 1024

typedef double matrix[MAX_SIZE * MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan MultiProcessed\n");
    int i, timestart, timeend, iter;

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/
    work();
    if (PRINT == 1)
        Print_Matrix();
}

__global__ void
division_step(double* d_A, double* d_b, double* d_y, int N, int k) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure j is within bounds for the current row k
    if (j > k && j < N) {
        d_A[k * N + j] /= d_A[k * N + k];  // Division step for row k
    }

    __syncthreads(); // Ensure all threads in the block finish before proceeding

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_y[k] = d_b[k] / d_A[k * N + k];
        d_A[k * N + k] = 1.0; // Set the diagonal element to 1.0
    }
}


__global__ void
elimination_step(double* d_A, double* d_b, double* d_y, int N, int k){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / N;
    int j = index % N;

    if ((i > k && i < N)&&(j > k && j < N)) {
        d_A[i * N + j] -= (d_A[i * N + k] * d_A[k * N + j]);  // Elimination
    }
    __syncthreads(); // Ensure all threads in the block finish before proceeding
    if (i > k && i < N && j == k) {
        d_b[i] -= d_A[i * N + k] * d_y[k];
        d_A[i * N + k] = 0.0;
    }
}


__global__ void
additional_step(double* d_A, double* d_b, double* d_y, int N, int k) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / N;
    int j = index % N;
    if (i < k && j > k && j < N) {
        d_A[i * N + j] -= (d_A[i * N + k] * d_A[k * N + j]);  // Additional Elimination
    }
    __syncthreads(); // Ensure all threads in the block finish before proceeding
    if (i < k && j == k) {
        d_y[i] -= d_A[i * N + k] * d_y[k];
        d_A[i * N + k] = 0.0;
    }
}




void
work(void)
{
    double* d_A;
    double* d_b;
    double* d_y;
    int k;
    int nr_blocks = (N*N + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;

    // Allocate memory on device
    hipMalloc((void**)&d_A, N * N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));

    // Copy data to device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(double), hipMemcpyHostToDevice);

    // Call the kernel
    for (k = 0; k < N; k++) {
        division_step<<<nr_blocks, THREADS_PER_BLOCK>>>(d_A, d_b, d_y, N, k);
        hipDeviceSynchronize();

        elimination_step<<<nr_blocks, THREADS_PER_BLOCK>>>(d_A, d_b, d_y, N, k);
        hipDeviceSynchronize();

        additional_step<<<nr_blocks, THREADS_PER_BLOCK>>>(d_A, d_b, d_y, N, k);
        hipDeviceSynchronize();
    }
    // Copy data back to host
    hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_y);
}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i * N + j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i * N + j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i * N + j] = 5.0;
                else
                    A[i * N + j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i * N + j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}
