/***************************************************************************
 *
 * Paralell version of Gauss-Jordan row reduction
 *
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_SIZE 4096

typedef double matrix[MAX_SIZE * MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* forward declarations */
void work(double*, double*, double*);
void Init_Matrix(void);
void Print_Matrix(void);
void Print_d_Matrix(double*, double*);
void Init_Default(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan\n");

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/

    double* d_A;
    double* d_b;
    double* d_y;


    // Allocate memory on device
    hipMalloc((void**)&d_A, N * N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));

    // Copy data to device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(double), hipMemcpyHostToDevice);

    work(d_A, d_b, d_y);

    // Copy data back to host
    hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_y);

    if (PRINT == 1)
        Print_Matrix();
}


__global__ void
division_step(double* d_A, double pivot, double* d_b, double* d_y, int N, int k) {
    int j = blockIdx.x * blockDim.x + threadIdx.x; 

    if (j != k && j < N) {
        d_A[k * N + j] = d_A[k * N + j] / d_A[k * N + k];
        // printf("d_y=%f, d_b[k]=%f, d_A[k * N + k]=%f\n", d_y[k], d_b[k], d_A[k * N + k]);
        d_y[k] = d_b[k] / pivot;
        d_A[k * N + k] = 1.0;
    }
}

__global__ void
under_elimination(double* d_A, double* d_b, double* d_y, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Row
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Column

    if (i < N && j < N && i >= k && j >= k) {
        printf("i=%d, j=%d, N=%d \n", i, j, N);
        d_A[i * N + j] = d_A[i * N + j] - d_A[i * N + k] * d_A[k * N + j];
        printf("d_A[i * N + j]=%f, d_A[i * N + k]=%f, d_A[k * N + j]=%f\n", d_A[i * N + j], d_A[i * N + k], d_A[k * N + j]);
        d_b[i] = d_b[i] - d_A[i * N + k] * d_y[k];
        d_A[i * N + k] = 0.0;
    }

}

__global__ void
upper_elimination(double* d_A, double* d_b, double* d_y, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Row
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Column

    if (i < N && j < N && i < k) {
        d_A[i * N + j] = d_A[i * N + j] - d_A[i * N + k] * d_A[k * N + j];
        d_b[i] = d_b[i] - d_A[i * N + k] * d_y[k];
        d_A[i * N + k] = 0.0;
    }
}


void
work(double* d_A, double* d_b, double* d_y)
{
    int i, j, k;

    int blockSize = 16;
    dim3 blockShape = dim3(blockSize, blockSize);
    dim3 gridShape = dim3((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    /* Gaussian elimination algorithm, Algo 8.4 from Grama */
    for (k = 0; k < N; k++) { /* Outer loop */
        double pivot;
        hipMemcpy(&pivot, &d_A[k * N + k], sizeof(double), hipMemcpyDeviceToHost);
        division_step<<<gridShape, blockShape>>>(d_A, pivot, d_b, d_y, N, k);
        hipDeviceSynchronize();
        // d_y[k] = b[k] / d_A[k * N + k];
        // printf("d_y: %f\n", y[k]);
        printf("Division step\n");
        Print_d_Matrix(d_A, d_y);

        under_elimination<<<gridShape, blockShape>>>(d_A, d_b, d_y, N, k);
        hipDeviceSynchronize();
        printf("Under elimination\n");
        Print_d_Matrix(d_A, d_y);

        upper_elimination<<<gridShape, blockShape>>>(d_A, d_b, d_y, N, k);
        hipDeviceSynchronize();
        printf("Upper elimination\n");
        Print_d_Matrix(d_A, d_y);

    }
}



void
Init_Matrix()
{
    int i, j;

    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i * N + j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i * N + j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i * N + j] = 5.0;
                else
                    A[i * N + j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i,j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i * N + j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Print_d_Matrix(double* d_A, double* d_y){
    int i, j;
    double* print_A = (double*)malloc(N * N * sizeof(double));
    double* print_y = (double*)malloc(N * sizeof(double));
    hipMemcpy(print_A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(print_y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    printf("Matrix d_A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", print_A[i * N + j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", print_y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}
